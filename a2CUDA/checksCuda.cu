#include "chckfnsCuda.cuh"
// CUDA header file
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#define ORDER 9
__device__ extern int sudoku[ORDER][ORDER];
__device__ __host__ int checkRow(int row, int num)
{
	int foundnum = 0;
	for (int i = 0; i < 9; i++) {
		if (sudoku[row][i] == num) {
			foundnum = 1;
			break;
		}
	}
	if (foundnum == 1)
		return 0;
	else
		return 1;
}

__device__ __host__ int checkColumn(int column, int num)
{
	int foundnum = 0;
	for (int i = 0; i < 9; i++) {
		if (sudoku[i][column] == num) {
			foundnum = 1;
			break;
		}
	}
	if (foundnum == 1)
		return 0;
	else
		return 1;
}

__device__ __host__ int checkSquare(int row, int column, int num)
{
	int foundnum = 0;
	int startIndexRow = (row / 3) * 3;
	int startIndexColumn = (column / 3) * 3;
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			if (sudoku[startIndexRow + i][startIndexColumn + j] == num) {
				foundnum = 1;
				break;
			}
		}
	}
	if (foundnum == 1)
		return 0;
	else
		return 1;
}
