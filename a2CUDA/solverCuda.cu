#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "chckfnsCuda.cuh"
#include <stdio.h>

#include <iostream>
#include <fstream>
#include <chrono>
#define ORDER 9


using namespace std;

const int ntpb = 9;  //9 threads per blocks

int sudoku[ORDER][ORDER] = { 0 };
int isClueGiven[ORDER][ORDER] = { 0 };
int prevPosition[ORDER][ORDER][2];
int placeNum(int row, int column);
void reportTime(const char* msg, chrono::steady_clock::duration span);

void print(int matrix[ORDER][ORDER]) //host code use only
{
	for (int i = 0; i < ORDER; i++) {
		for (int j = 0; j < ORDER; j++)
			cout << matrix[i][j] << " ";
		cout << endl;
	}

	cout << endl;
	return;
}

//kernel 1 - store position

__global__ void storePositions() //kernel
{
	int temprow, tempcolumn;
	temprow = -1;
	tempcolumn = -1;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	if (isClueGiven[x][y] == 0) {
		prevPosition[x][y][0] = temprow;
		prevPosition[x][y][1] = tempcolumn;
		temprow = x;
		tempcolumn = y;
	}	
}
__device__ __host__ int goBack(int &row, int &column)
{
	int trow, tcolumn;

	if (row == 0 && column == 0)
		return 0;
	sudoku[row][column] = 0;

	trow = prevPosition[row][column][0];
	tcolumn = prevPosition[row][column][1];
	tcolumn -= 1;

	row = trow;
	column = tcolumn;

	return 1;
}
__device__ __host__ int placeNum(int row, int column)
{
	if (isClueGiven[row][column] == 1)
		return 1;

	for (int num = sudoku[row][column] + 1; num <= 9; num++) {
		if (checkRow(row, num) && checkColumn(column, num) && checkSquare(row, column, num)) {
			sudoku[row][column] = num;
			return 1;
		}
	}

	sudoku[row][column] = 0;

	return 0;
}


__device__ __host__ int solveSudoku(){

	for (int row = 0; row < 9; row++) {
		for (int column = 0; column < 9; column++) {
			if (!placeNum(row, column)) {
				sudoku[row][column] = 0;
				if (!goBack(row, column))
					return 0;
			}
		}
	}
	return 1;
}


int main(int argc, char* argv[])
{
	fstream file;
	chrono::steady_clock::time_point ts, te;

	int nblks = 9;  // hard coded makes sense right? It can only have 9 blocks.


	if (argc == 2)
	{
		file.open(argv[1], ios::in);

		if (file.is_open())
		{
			for (int row = 0; row < ORDER; row++) {
				for (int column = 0; column < ORDER; column++) {
					file >> sudoku[row][column];
					if (sudoku[row][column] != 0)
						isClueGiven[row][column] = 1;
				}
			}
			print(sudoku);
		}
		else
			cout << "Could not locate file ' " << argv[1] << "'. Enter elements manually" << endl;
	}

	if (argc > 2)
		cout << "More than one arguments. Enter elements manually\n";

	if (!file.is_open()) {
		cout << "Enter 81  elements (0s for cells without clues) :" << endl;

		for (int row = 0; row < ORDER; row++) {
			for (int column = 0; column < ORDER; column++) {
				cin >> sudoku[row][column];
				if (sudoku[row][column] != 0)
					isClueGiven[row][column] = 1;
			}
		}

		print(sudoku);
	}

	ts = chrono::steady_clock::now();
	storePositions();
	te = chrono::steady_clock::now();
	reportTime("Position storage time:", te - ts);
	ts = chrono::steady_clock::now();
	solveSudoku();
	te = chrono::steady_clock::now();
	reportTime("Time to solve:", te - ts);
	print(sudoku);
	return 0;
}
// report system time
//
void reportTime(const char* msg, chrono::steady_clock::duration span) {
	auto ms = chrono::duration_cast<chrono::milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
	std::cout << "" << std::endl;
}